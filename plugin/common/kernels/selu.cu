#include "hip/hip_runtime.h"
#include "kernel.h"

template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA) __global__
    void seluKernel(const int n, const float lambd, const float* input, float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        float scale = 1.05f;
        float alpha = 1.67326f;

        output[i] = scale * (max(0.0f, input[i]) + min(0.0f, alpha * (exp(input[i]) - 1)));
    }
}

pluginStatus_t seluGPU(hipStream_t stream, const int n, const float lambd, const void* input, void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    seluKernel<BS><<<GS, BS, 0, stream>>>(n, lambd,
                                           (const float*) input,
                                           (float*) output);
    return STATUS_SUCCESS;
}

pluginStatus_t seluInference(
    hipStream_t stream, const int n, const float lambd, const void* input, void* output)
{
    return seluGPU(stream, n, lambd, (const float*) input, (float*) output);
}
